#include <iostream>
#include <vector>
#include <string>
#include <sstream>
#include <fstream>
#include <cmath>
#include <numeric>
#include <algorithm>
#include <ctime>
#include <random>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#define CUDA_CHECK(err) do {     if (err != hipSuccess) {         cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << endl;         exit(1);     } } while(0)

vector<string> split(const string& s, char delimiter) {
    vector<string> tokens;
    string token;
    stringstream ss(s);
    while (getline(ss, token, delimiter)) tokens.push_back(token);
    return tokens;
}

vector<double> readCSV(const string& filename) {
    ifstream file(filename);
    vector<double> prices;
    string line;

    if (!file.is_open()) {
        cerr << "Error: Could not open file " << filename << endl;
        return prices;
    }

    for (int i = 0; i < 3 && getline(file, line); ++i) {}

    while (getline(file, line)) {
        vector<string> tokens = split(line, ',');
        if (tokens.size() < 2) continue;
        try {
            prices.push_back(stod(tokens[1]));
        } catch (...) {
            continue;
        }
    }
    return prices;
}

__global__ void monteCarloKernel(double* finalPrices, double lastPrice, double mu, double sigma, int days, int simulations, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= simulations) return;

    hiprandState state;
    hiprand_init(seed + idx, 0, 0, &state); 

    double price = lastPrice;
    double dt = 1.0;
    for (int d = 0; d < days; d++) {
        double z = hiprand_normal_double(&state); 
        double drift = (mu - 0.5 * sigma * sigma) * dt;
        double shock = sigma * sqrt(dt) * z;
        price *= exp(drift + shock);
    }
    finalPrices[idx] = price;
}

int main() {
    clock_t start_time = clock();

    string filename = "aapl_stock_data.csv";
    vector<double> prices = readCSV(filename);

    if (prices.size() < 60) {
        cerr << "Not enough price data! Need at least 60 data points." << endl;
        return 1;
    }

    vector<double> logReturns;
    for (size_t i = 1; i < prices.size(); i++) {
        if (prices[i - 1] != 0) {
            double r = log(prices[i] / prices[i - 1]);
            logReturns.push_back(r);
        }
    }

    if (logReturns.empty()) {
        cerr << "No valid log returns calculated!" << endl;
        return 1;
    }

    double mean_r = accumulate(logReturns.begin(), logReturns.end(), 0.0) / logReturns.size();
    double var_r = 0.0;
    for (double r : logReturns) var_r += (r - mean_r) * (r - mean_r);
    var_r /= (logReturns.size() > 1 ? logReturns.size() - 1 : 1);
    double sigma_r = sqrt(var_r);

    vector<double> truncatedReturns;
    for (double r : logReturns) {
        if (fabs(r - mean_r) <= 3 * sigma_r) {
            truncatedReturns.push_back(r);
        }
    }

    if (truncatedReturns.empty()) {
        cerr << "No valid truncated returns calculated!" << endl;
        return 1;
    }

    int window = min(30, (int)truncatedReturns.size());
    double rollingMean = accumulate(truncatedReturns.end() - window, truncatedReturns.end(), 0.0) / window;

    double sigma = sigma_r;
    double mu = rollingMean + 0.5 * sigma * sigma;

    int days = 30;
    int simulations = 20000;

    double lastPrice = prices[prices.size() - days - 1];
    double actualFuturePrice = prices.back();

    double* d_finalPrices;
    CUDA_CHECK(hipMalloc(&d_finalPrices, simulations * sizeof(double)));

    int threadsPerBlock = 256;
    int blocks = (simulations + threadsPerBlock - 1) / threadsPerBlock;

    random_device rd;
    unsigned long long seed = rd();
    monteCarloKernel<<<blocks, threadsPerBlock>>>(d_finalPrices, lastPrice, mu, sigma, days, simulations, seed);
    CUDA_CHECK(hipDeviceSynchronize());

    vector<double> finalPrices(simulations);
    CUDA_CHECK(hipMemcpy(finalPrices.data(), d_finalPrices, simulations * sizeof(double), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_finalPrices));


    double avg = accumulate(finalPrices.begin(), finalPrices.end(), 0.0) / finalPrices.size();
    sort(finalPrices.begin(), finalPrices.end());
    double median = finalPrices[finalPrices.size() / 2];
    double p5 = finalPrices[finalPrices.size() * 0.05];
    double p95 = finalPrices[finalPrices.size() * 0.95];
    double minP = *min_element(finalPrices.begin(), finalPrices.end());
    double maxP = *max_element(finalPrices.begin(), finalPrices.end());

    double mae = fabs(median - actualFuturePrice);
    double mape = fabs((median - actualFuturePrice) / actualFuturePrice) * 100.0;
    bool covered = (actualFuturePrice >= p5 && actualFuturePrice <= p95);

    cout << fixed << setprecision(2);
    cout << "=== GBM + Monte Carlo Prediction - CUDA (30 days) ===" << endl;
    cout << "Start Price (30 days prior): " << lastPrice << endl;
    cout << "Estimated Drift (mu): " << mu << endl;
    cout << "Estimated Volatility (sigma): " << sigma << endl;
    cout << "Predicted Average Price: " << avg << endl;
    cout << "Median Predicted Price: " << median << endl;
    cout << "5th Percentile: " << p5 << " | 95th Percentile: " << p95 << endl;
    cout << "Min Predicted Price: " << minP << " | Max Predicted Price: " << maxP << endl;
    cout << "-----------------------------------" << endl;
    cout << "Actual Price after " << days << " days: " << actualFuturePrice << endl;
    cout << "MAE (Median vs Actual): " << mae << endl;
    cout << "MAPE (Median vs Actual): " << mape << "%" << endl;
    cout << "Coverage (Actual price inside 5%-95% interval): " << (covered ? "YES" : "NO") << endl;

    clock_t end_time = clock();
    cout << "Execution Time: " << double(end_time - start_time) / CLOCKS_PER_SEC << " seconds" << endl;

    return 0;
}